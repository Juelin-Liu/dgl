#include "hip/hip_runtime.h"
//
// Created by juelinliu on 12/11/23.
//

#include "array_ops.cuh"
#include "../../runtime/cuda/cuda_common.h"
#include <cstdint>
#include <hipcub/hipcub.hpp>
#include "../../array/cuda/atomic.cuh"

namespace dgl::dev
{
    namespace impl 
    {
        /**
            * @brief Compute the size of each row in the sampled CSR, without replacement.
            *
            * @tparam IdType The type of node and edge indexes.
            * @param num_rows The number of rows to pick.
            * @param in_rows The set of rows to pick.
            * @param dflag The mask indicating the presense of number in row
            * `in_rows` (output).
            */
        template<typename IdType>
        __global__ void _Mask(const IdType * in_row, const int64_t num_rows, int8_t* dflag) {
            const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
            if (tIdx < num_rows) {
                dflag[ in_row[tIdx] ] = 1;
            }
        }

        template<typename IdType>
        __global__ void Increment(IdType * array, const int64_t array_len, const IdType * row, const int64_t num_item) {
            const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
            if (tIdx < num_item) {
                const IdType rIdx = row[tIdx];
                aten::cuda::AtomicAdd(array + rIdx, static_cast<IdType>(1));
            }
        }

        template<typename CounterType, typename IndexType>
        __global__ void Increment(CounterType * array, const int64_t array_len, 
                                const IndexType * row, const int64_t num_item) {
            const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
            if (tIdx < num_item) {
                const IndexType rIdx = row[tIdx];
                aten::cuda::AtomicAdd(array + rIdx, static_cast<CounterType>(1));
            }
        }
    } // impl



    void Reset(NDArray& bitmap){
//            auto stream = runtime::getCurrentCUDAStream();
//        CUDA_CALL(hipMemsetAsync(bitmap.Ptr<int8_t>(), 0, capacity, stream));
            CUDA_CALL(hipMemset(bitmap.Ptr<int8_t>(), 0, bitmap.GetSize()));
    };


    int64_t NumItem(const NDArray& bitmap){
        auto ctx = bitmap->ctx;
        auto device = runtime::DeviceAPI::Get(ctx);
        int64_t *d_num_item = static_cast<int64_t*>(device->AllocWorkspace(ctx, sizeof(int64_t)));
        int64_t h_num_item = 0;
        size_t temp_storage_bytes = 0;
        auto stream = runtime::getCurrentCUDAStream();
        const int8_t * dflag = bitmap.Ptr<const int8_t>();
        const int64_t num_items = bitmap.NumElements();
        CUDA_CALL(hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes, dflag, d_num_item, num_items, stream));
        void * d_temp_storage = device->AllocWorkspace(ctx, temp_storage_bytes);
        CUDA_CALL(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, dflag, d_num_item, num_items, stream));
        CUDA_CALL(hipMemcpyAsync(&h_num_item, d_num_item, sizeof(int64_t), hipMemcpyDeviceToHost, stream));
        device->StreamSync(ctx, stream);
        device->FreeWorkspace(ctx, d_temp_storage);
        device->FreeWorkspace(ctx, d_num_item);
        return h_num_item;
    };

    template<DGLDeviceType XPU, typename IdType>
    void Mask(NDArray& bitmap, NDArray row){
        const dim3 block(256);
        const dim3 grid((row.NumElements() + block.x - 1) / block.x);
        auto stream = runtime::getCurrentCUDAStream();
        int8_t * dflag = bitmap.Ptr<int8_t>();
        CUDA_KERNEL_CALL(impl::_Mask<IdType>, grid, block, 0, stream, static_cast<IdType* >(row->data), row.NumElements(), dflag);
    };

    template<DGLDeviceType XPU, typename IdType>
    NDArray Flagged(const NDArray& bitmap, DGLContext ctx){
        auto device = runtime::DeviceAPI::Get(ctx);
        auto stream = runtime::getCurrentCUDAStream();
        int64_t *d_num_item = static_cast<int64_t*>(device->AllocWorkspace(ctx, sizeof(int64_t)));
        int64_t num_item = NumItem(bitmap);
        int64_t v_num = bitmap.NumElements();
        auto d_in = hipcub::CountingInputIterator<IdType>(0);
        NDArray flagged = NDArray::Empty({num_item}, DGLDataTypeTraits<IdType>::dtype, ctx);
        IdType *d_out = static_cast<IdType*>(flagged->data);
        size_t   temp_storage_bytes = 0;
        const int8_t * dflag = bitmap.Ptr<const int8_t>();
        hipcub::DeviceSelect::Flagged(nullptr, temp_storage_bytes, d_in, dflag, d_out, d_num_item, v_num, stream);
        void     *d_temp_storage = device->AllocWorkspace(ctx, temp_storage_bytes);
        hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_in, dflag, d_out, d_num_item, v_num, stream);

        device->StreamSync(ctx, stream);
        device->FreeWorkspace(ctx, d_temp_storage);
        device->FreeWorkspace(ctx, d_num_item);
        return flagged;
    };
    
    template<DGLDeviceType XPU, typename IdType>
    void Increment(NDArray& count, const NDArray& row){
        CHECK_EQ(count->ctx, row->ctx);
        const dim3 block(256);
        const dim3 grid((row.NumElements() + block.x - 1) / block.x);
        auto stream = runtime::getCurrentCUDAStream();
        CUDA_KERNEL_CALL(impl::Increment, grid, block, 0, stream, count.Ptr<IdType>(), count.NumElements(), row.Ptr<IdType>(), row.NumElements());
        auto device = runtime::DeviceAPI::Get(count->ctx);
        device->StreamSync(count->ctx, stream);
    }; 

    template<DGLDeviceType XPU, typename CounterType, typename IndexType>
    void Increment(NDArray& count, const NDArray& row){
        CHECK_EQ(count->ctx, row->ctx);
        const dim3 block(256);
        const dim3 grid((row.NumElements() + block.x - 1) / block.x);
        auto stream = runtime::getCurrentCUDAStream();
        CUDA_KERNEL_CALL(impl::Increment, grid, block, 0, stream, count.Ptr<CounterType>(), count.NumElements(), row.Ptr<IndexType>(), row.NumElements());
        auto device = runtime::DeviceAPI::Get(count->ctx);
        device->StreamSync(count->ctx, stream);
    }; 

    template void Mask<kDGLCUDA, int32_t>(NDArray&, NDArray);
    template void Mask<kDGLCUDA, int64_t>(NDArray&, NDArray);
    template NDArray Flagged<kDGLCUDA, int32_t>(const NDArray&, DGLContext);
    template NDArray Flagged<kDGLCUDA, int64_t>(const NDArray&, DGLContext);
    template void Increment<kDGLCUDA, int32_t>(NDArray&, const NDArray&);
    template void Increment<kDGLCUDA, int64_t>(NDArray&, const NDArray&);
    template void Increment<kDGLCUDA, int32_t, int32_t>(NDArray&, const NDArray&);
    template void Increment<kDGLCUDA, int64_t, int32_t>(NDArray&, const NDArray&);
    template void Increment<kDGLCUDA, int32_t, int64_t>(NDArray&, const NDArray&);
    template void Increment<kDGLCUDA, int64_t, int64_t>(NDArray&, const NDArray&);
}