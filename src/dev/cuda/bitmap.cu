#include "hip/hip_runtime.h"
//
// Created by juelin on 2/6/24.
//
#include <hipcub/hipcub.hpp>

#include "../../runtime/cuda/cuda_common.h"
#include "bitmap.h"

namespace dgl::dev {
class DeviceBitIterator
    : public std::iterator<std::random_access_iterator_tag, bool> {
 private:
  Bucket* _bitmap{nullptr};
  int64_t _offset{0};
  int64_t _num_buckets{0};

 public:
  using self_type = DeviceBitIterator;
  using value_type = bool;

  __host__ __device__
  DeviceBitIterator(Bucket* bitmap, int64_t num_buckets, int64_t offset = 0)
      : _bitmap{bitmap},
        _offset{offset},
        _num_buckets{num_buckets} {
            // printf("Constructor offset = %ld\n", _offset);
        };

  __host__ __device__ __forceinline__ self_type operator++(int) {
    self_type retval = *this;
    _offset++;
    // printf("operator++(int) offset = %ld\n", _offset);

    return retval;
  }

  __host__ __device__ __forceinline__ self_type operator++() {
    _offset++;
    // printf("operator++() offset = %ld\n", _offset);

    return *this;
  }

  __host__ __device__ __forceinline__ bool operator*() const {
    const uint32_t bucket_idx = _offset / (sizeof(Bucket) * 8);
    const Bucket shift = _offset % (sizeof(Bucket) * 8);
    const Bucket mask = 1u << shift;
    const Bucket flag = _bitmap[bucket_idx];
    bool retval = (flag & mask);
    // printf("operator*() flag %d shift %d mask %d offset %ld ret %d\n", flag,
    // shift, mask, _offset, retval);
    return retval;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator+(Distance n) const {
    self_type retval(_bitmap, _num_buckets, _offset + n);
    // printf("operator+(Distance n) offset = %ld n=%d \n", _offset, n);

    return retval;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator+=(Distance n) const {
    _offset += n;
    // printf("operator+=(Distance n) offset = %ld n=%d \n", _offset, n);

    return *this;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator-(Distance n) const {
    self_type retval(_bitmap, _num_buckets, _offset - n);
    // printf("operator-(Distance n) offset = %ld n=%d \n", _offset, n);

    return retval;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator-=(Distance n) const {
    _offset -= n;
    // printf("operator-=(Distance n) offset = %ld n=%d \n", _offset, n);

    return *this;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ value_type operator[](Distance n) const {
    const Distance bucket_idx = (_offset + n) / (sizeof(Bucket) * 8);
    const Bucket shift = (_offset + n) % (sizeof(Bucket) * 8);
    const Bucket mask = 1u << shift;
    const Bucket flag = _bitmap[bucket_idx];
    bool retval = (flag & mask);
    // printf("operator[] n %d flag %d shift %d mask %d offset %ld ret %d\n", n,
    // flag, shift, mask, _offset, retval);
    return retval;
  }

  template <typename Distance>
  __device__ __forceinline__ void flag(Distance n) {
    assert(n < _num_buckets * sizeof(Bucket) * 8);
    const Distance bucket_idx = (_offset + n) / (sizeof(Bucket) * 8);
    const Bucket shift = (_offset + n) % (sizeof(Bucket) * 8);
    const Bucket mask = 1u << shift;
    atomicOr(_bitmap + bucket_idx, mask);
    // printf("flag n %ld shift %d mask %d offset %ld bitmap %d\n", n, shift,
    // mask, _offset, _bitmap[bucket_idx]);
  }

  template <typename Distance>
  __device__ __forceinline__ Bucket popcnt(Distance bucket_idx) const {
    return __popc(_bitmap[bucket_idx]);
  }

  template <typename Distance>
  __device__ __forceinline__ Bucket
  popcnt(Distance bucket_idx, Distance num_bits) const {
    const int shift = 8 * sizeof(Bucket) - num_bits;
    const Bucket bitmap = _bitmap[bucket_idx];
    const Bucket mask = bitmap << shift;
    const int retval = __popc(mask);
    // printf("popcnt bucket_idx %ld num_bits %ld retval %d bitmap %d shift %d
    // mask %d\n", bucket_idx, num_bits, retval, bitmap, shift, mask);
    return retval;
  }
  __host__ __device__ __forceinline__ bool operator==(
      const self_type& rhs) const {
    return _bitmap == rhs._bitmap && _offset == rhs._offset &&
           _num_buckets == rhs._num_buckets;
  }

  __host__ __device__ __forceinline__ bool operator!=(
      const self_type& rhs) const {
    return _bitmap != rhs._bitmap || _offset != rhs._offset ||
           _num_buckets == rhs._num_buckets;
  }
};

namespace impl {
template <typename IdType>
__global__ void flag_kernel(
    DeviceBitIterator iter, const IdType *row, int64_t num_rows) {
  const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tIdx < num_rows) {
    iter.flag(row[tIdx]);
  }
}

// assume d_num_item_out is 0
__global__ void cnt_kernel(DeviceBitIterator iter, int64_t num_buckets, uint32_t * d_num_item_out) {
  const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  typedef hipcub::BlockReduce<uint32_t, 256> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  uint32_t thread_data{0};
  if (tIdx < num_buckets) {
    thread_data = iter.popcnt(tIdx);
  }
  uint32_t aggregate = BlockReduce(temp_storage).Sum(thread_data);
  if (threadIdx.x == 0) {
    atomicAdd(d_num_item_out, aggregate);
  }
}

__global__ void popcnt_kernel(
    DeviceBitIterator iter, int64_t num_buckets, Offset *offset) {
  const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tIdx < num_buckets) {
    offset[tIdx] = iter.popcnt(tIdx);
  }
}

template <typename IdType>
__global__ void map_kernel(
    DeviceBitIterator iter, const Offset *offset, const IdType *row,
    int64_t num_rows, IdType *out_row) {
  const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tIdx < num_rows) {
    const IdType id = row[tIdx];
    assert(iter[id] == true);
    const int64_t bucket_idx = id / (8 * sizeof(Bucket));
    const int64_t num_bits = id % (8 * sizeof(Bucket)) + 1;
    const IdType loc_id =
        offset[bucket_idx] + iter.popcnt(bucket_idx, num_bits);
    out_row[tIdx] = loc_id;
  }
}

__global__ void read_kernel(DeviceBitIterator iter, int64_t num_items) {
  const int64_t tIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (tIdx < num_items) {
    const int flag = iter[tIdx];
    printf("tIdx: %lld mask %d\n", tIdx, flag);
  }
}
}  // namespace impl

DeviceBitmap::DeviceBitmap(int64_t num_elems, DGLContext ctx, bool allow_remap) {
  _allow_remap = allow_remap;
  int64_t bucket_bits = sizeof(Bucket) * 8;
  _num_buckets = (num_elems + bucket_bits - 1) / bucket_bits;  // 32 bits per buckets
  _ctx = ctx;
  auto device = runtime::DeviceAPI::Get(_ctx);
  auto stream = runtime::getCurrentCUDAStream();

  _bitmap = static_cast<Bucket *>(device->AllocWorkspace(ctx, _num_buckets * sizeof(Bucket)));
  CUDA_CALL(hipMemsetAsync(&_bitmap, 0, _num_buckets * sizeof(Bucket), stream));
  if (_allow_remap) {
    _offset = static_cast<Bucket *>(device->AllocWorkspace(ctx, (_num_buckets + 1) * sizeof(Offset)));
    CUDA_CALL(hipMemsetAsync(&_offset, 0, (_num_buckets + 1) * sizeof(Bucket), stream));
  }
}

DeviceBitmap::~DeviceBitmap() {
//  if (_bitmap) hipFree(_bitmap);
//  if (_offset) hipFree(_offset);
  auto device = runtime::DeviceAPI::Get(_ctx);

  if (_bitmap) device->FreeWorkspace(_ctx, _bitmap);
  if (_offset) device->FreeWorkspace(_ctx, _offset);
}

void DeviceBitmap::reset() {
  auto device = runtime::DeviceAPI::Get(_ctx);
  auto stream = runtime::getCurrentCUDAStream();
  hipMemsetAsync(&_bitmap, 0, _num_buckets * sizeof(Bucket), stream);
  if(_allow_remap) hipMemsetAsync(&_offset, 0, (_num_buckets + 1) * sizeof(Bucket), stream);
  device->StreamSync(_ctx, stream);
  _num_flagged = 0;
}

template <typename IdType>
void DeviceBitmap::flag(const IdType *row, int64_t num_rows) {
  auto stream = runtime::getCurrentCUDAStream();
  auto device = runtime::DeviceAPI::Get(_ctx);

  const dim3 block(256);
  const dim3 grid((num_rows + block.x - 1) / block.x);
  DeviceBitIterator iter(_bitmap, _num_buckets, 0);
  CUDA_KERNEL_CALL(impl::flag_kernel, grid, block, 0, stream, iter, row, num_rows);
//  impl::flag_kernel<<<grid, block, 0>>>(iter, row, num_rows);
//  device->StreamSync(_ctx, stream);
  _build_map = false;
}


int64_t DeviceBitmap::buildMap() {
  CHECK(_allow_remap);
  auto device = runtime::DeviceAPI::Get(_ctx);
  auto stream = runtime::getCurrentCUDAStream();
  const dim3 block(256);
  const dim3 grid((_num_buckets + block.x - 1) / block.x);
  DeviceBitIterator iter(_bitmap, _num_buckets, 0);
  CUDA_KERNEL_CALL(impl::popcnt_kernel, grid, block, 0, stream, iter, _num_buckets, _offset);
//  impl::popcnt_kernel<<<grid, block, 0>>>(iter, _num_buckets, _offset);

  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  auto d_in = _offset;
  auto d_out = _offset;
  auto num_items = _num_buckets + 1;
  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream));
//  hipMalloc(&d_temp_storage, temp_storage_bytes);
  d_temp_storage = device->AllocWorkspace(_ctx, temp_storage_bytes);

  CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
      d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream));
  CUDA_CALL(hipMemcpyAsync(
      &_num_flagged, _offset + _num_buckets, sizeof(Offset), hipMemcpyDefault, stream));

  device->StreamSync(_ctx, stream);
  device->FreeWorkspace(_ctx, d_temp_storage);
  _build_map = true;
  return _num_flagged;
}

int64_t DeviceBitmap::numItem() const {
  auto device = runtime::DeviceAPI::Get(_ctx);
  auto stream = runtime::getCurrentCUDAStream();
  const dim3 block(256);
  const dim3 grid((_num_buckets + block.x - 1) / block.x);
  DeviceBitIterator iter(_bitmap, _num_buckets, 0);
  auto * d_num_item = static_cast<uint32_t*>(device->AllocWorkspace(_ctx, sizeof(uint32_t)));
  uint32_t h_num_item{0};
  CUDA_CALL(hipMemsetAsync(d_num_item, 0, sizeof(uint32_t), stream));
  CUDA_KERNEL_CALL(impl::cnt_kernel, grid, block, 0, stream, iter, _num_buckets, d_num_item);
  CUDA_CALL(hipMemcpyAsync(&h_num_item, d_num_item, sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
  device->StreamSync(_ctx, stream);
  return h_num_item;
}

template <typename IdType>
int64_t DeviceBitmap::unique(IdType *out_row) const {
  auto device = runtime::DeviceAPI::Get(_ctx);
  auto stream = runtime::getCurrentCUDAStream();
  // adopted from cub
  // Declare, allocate, and initialize device-accessible pointers for input,
  // flags, and output
  int num_items = _num_buckets * sizeof(Bucket) * 8;
  auto d_in =
      hipcub::CountingInputIterator<IdType>(0);  // e.g., [1, 2, 3, 4, 5, 6, 7, 8]
  DeviceBitIterator d_flags(
      _bitmap, _num_buckets, 0);  // e.g., [1, 0, 0, 1, 0, 1, 1, 0]
  IdType *d_out = out_row;        // e.g., [ ,  ,  ,  ,  ,  ,  ,  ]
  int64_t *d_num_selected_out = static_cast<int64_t *>(device->AllocWorkspace(_ctx, sizeof(int64_t)));
//  hipMalloc(&d_num_selected_out, sizeof(int64_t));
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  CUDA_CALL(hipcub::DeviceSelect::Flagged(
      d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out,
      d_num_selected_out, num_items, stream));
  device->StreamSync(_ctx, stream);
  // Allocate temporary storage
//  hipMalloc(&d_temp_storage, temp_storage_bytes);
  d_temp_storage = device->AllocWorkspace(_ctx, temp_storage_bytes);
  // Run selection
  CUDA_CALL(hipcub::DeviceSelect::Flagged(
      d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out,
      d_num_selected_out, num_items, stream));
  // d_out                 <-- [1, 4, 6, 7]
  // d_num_selected_out    <-- [4]
  int64_t h_num_selected_out{0};
  hipDeviceSynchronize();
  CUDA_CALL(hipMemcpyAsync(
      &h_num_selected_out, d_num_selected_out, sizeof(int64_t),
      hipMemcpyDefault, stream));
//  hipFree(d_num_selected_out);
//  hipDeviceSynchronize();
  device->StreamSync(_ctx, stream);
  assert(h_num_selected_out == _num_flagged);
  device->FreeWorkspace(_ctx, d_temp_storage);
  device->FreeWorkspace(_ctx, d_num_selected_out);
  return h_num_selected_out;
};

template <typename IdType>
void DeviceBitmap::map(const IdType *row, int64_t num_rows, IdType *out_row) const {
  CHECK(_allow_remap && _build_map);
  auto device = runtime::DeviceAPI::Get(_ctx);
  auto stream = runtime::getCurrentCUDAStream();
  const dim3 block(256);
  const dim3 grid((num_rows + block.x - 1) / block.x);
  DeviceBitIterator iter(_bitmap, _num_buckets, 0);
  CUDA_KERNEL_CALL(impl::map_kernel, grid, block, 0, stream, iter, _offset, row, num_rows, out_row);
//  impl::map_kernel<<<grid, block, 0>>>(iter, _offset, row, num_rows, out_row);
//  hipDeviceSynchronize();
  device->StreamSync(_ctx, stream);
};

template void DeviceBitmap::flag<int32_t>(const int32_t *, int64_t);
template void DeviceBitmap::flag<int64_t>(const int64_t *, int64_t);

template int64_t DeviceBitmap::unique<int32_t>(int32_t *) const ;
template int64_t DeviceBitmap::unique<int64_t>(int64_t *) const ;

template void DeviceBitmap::map<int32_t>(const int32_t *, int64_t, int32_t *) const ;
template void DeviceBitmap::map<int64_t>(const int64_t *, int64_t, int64_t *) const ;
}  // namespace dgl::dev