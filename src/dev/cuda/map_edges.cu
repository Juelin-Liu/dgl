#include "hip/hip_runtime.h"
//
// Created by juelinliu on 12/11/23.
//

#include <assert.h>

#include "../../graph/unit_graph.h"
#include "../../runtime/cuda/cuda_common.h"
#include "../../runtime/cuda/cuda_hashtable.cuh"
#include "map_edges.cuh"
using namespace dgl::runtime::cuda;
namespace dgl::runtime::cuda {
template class OrderedHashTable<int32_t>;
template class OrderedHashTable<int64_t>;
}  // namespace dgl::runtime::cuda

namespace dgl::dev {
namespace impl {

template <typename IdType>
inline size_t RoundUpDiv(const IdType num, const size_t divisor) {
  return static_cast<IdType>(num / divisor) + (num % divisor == 0 ? 0 : 1);
}

template <typename IdType>
inline IdType RoundUp(const IdType num, const size_t unit) {
  return RoundUpDiv(num, unit) * unit;
}

template <typename IdType, int BLOCK_SIZE, IdType TILE_SIZE>
__device__ void map_vertex_ids(
    const IdType* const global, IdType* const new_global,
    const IdType num_vertices, const DeviceOrderedHashTable<IdType>& table) {
  assert(BLOCK_SIZE == blockDim.x);

  using Mapping = typename OrderedHashTable<IdType>::Mapping;

  const IdType tile_start = TILE_SIZE * blockIdx.x;
  const IdType tile_end = min(TILE_SIZE * (blockIdx.x + 1), num_vertices);

  for (IdType idx = threadIdx.x + tile_start; idx < tile_end;
       idx += BLOCK_SIZE) {
    const Mapping& mapping = *table.Search(global[idx]);
    new_global[idx] = mapping.local;
  }
}

/**
 * @brief Generate mapped edge endpoint ids.
 *
 * @tparam IdType The type of id.
 * @tparam BLOCK_SIZE The size of each thread block.
 * @tparam TILE_SIZE The number of edges to process per thread block.
 * @param global_srcs_device The source ids to map.
 * @param new_global_srcs_device The mapped source ids (output).
 * @param global_dsts_device The destination ids to map.
 * @param new_global_dsts_device The mapped destination ids (output).
 * @param num_edges The number of edges to map.
 * @param src_mapping The mapping of sources ids.
 * @param dst_mapping The mapping of destination ids.
 */
template <typename IdType, int BLOCK_SIZE, IdType TILE_SIZE>
__global__ void map_edge_ids(
    const IdType* const global_srcs_device,
    IdType* const new_global_srcs_device,
    const IdType* const global_dsts_device,
    IdType* const new_global_dsts_device, const IdType num_edges,
    DeviceOrderedHashTable<IdType> src_mapping,
    DeviceOrderedHashTable<IdType> dst_mapping) {
  assert(BLOCK_SIZE == blockDim.x);
  assert(2 == gridDim.y);

  if (blockIdx.y == 0) {
    map_vertex_ids<IdType, BLOCK_SIZE, TILE_SIZE>(
        global_srcs_device, new_global_srcs_device, num_edges, src_mapping);
  } else {
    map_vertex_ids<IdType, BLOCK_SIZE, TILE_SIZE>(
        global_dsts_device, new_global_dsts_device, num_edges, dst_mapping);
  }
}
}  // namespace impl
template <typename IdType>
void GPUMapEdges(aten::COOMatrix& mat, const OrderedHashTable<IdType>& hash) {
  constexpr const int BLOCK_SIZE = 128;
  constexpr const size_t TILE_SIZE = 1024;
  int64_t num_edges = mat.col.NumElements();
  const dim3 grid(impl::RoundUpDiv<IdType>(num_edges, TILE_SIZE), 2);
  const dim3 block(BLOCK_SIZE);
  auto stream = runtime::getCurrentCUDAStream();
  // map the srcs
  CUDA_KERNEL_CALL(
      (impl::map_edge_ids<IdType, BLOCK_SIZE, TILE_SIZE>), grid, block, 0,
      stream, mat.row.Ptr<IdType>(), mat.row.Ptr<IdType>(),
      mat.col.Ptr<IdType>(), mat.col.Ptr<IdType>(), num_edges,
      hash.DeviceHandle(), hash.DeviceHandle());
};
template void GPUMapEdges<int32_t>(
    aten::COOMatrix&, const OrderedHashTable<int32_t>&);
template void GPUMapEdges<int64_t>(
    aten::COOMatrix&, const OrderedHashTable<int64_t>&);
}  // namespace dgl::dev
